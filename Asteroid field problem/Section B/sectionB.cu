
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define MAX_CHAR 100
#define DATAFILE "data256.txt"
#define RESULTSFILE "resultsSectionB256.txt"
#define G 6.674e-11
#define NUM_ITER 20000
#define NUM_ITER_SHOW 1000

__global__ void calculateAsteroid(double * d_x, double * d_y, double * d_vx, double * d_vy, double * d_m) {
  int indexInteractuateAsteroid = blockIdx.x * blockDim.x + threadIdx.x;
  int indexInitialAsteroid = blockIdx.y * blockDim.y + threadIdx.y;

  if (indexInteractuateAsteroid != indexInitialAsteroid) {
    double d = sqrt(pow((d_x[indexInitialAsteroid] - d_x[indexInteractuateAsteroid]), 2.0) + pow((d_y[indexInitialAsteroid] - d_y[indexInteractuateAsteroid]), 2.0));
    double f = G * d_m[indexInitialAsteroid] * d_m[indexInteractuateAsteroid] / pow(d, 2.0);
    double fx = f * (d_x[indexInteractuateAsteroid] - d_x[indexInitialAsteroid]) / d;
    double ax = fx / d_m[indexInitialAsteroid];
    double fy = f * (d_y[indexInteractuateAsteroid] - d_y[indexInitialAsteroid]) / d;
    double ay = fy / d_m[indexInitialAsteroid];
    atomicAdd(&d_vx[indexInitialAsteroid], ax);
    atomicAdd(&d_vy[indexInitialAsteroid], ay);
  }
}
__global__ void updatePositions(double * d_x, double * d_y, double * d_vx, double * d_vy) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  d_x[i] += d_vx[i];
  d_y[i] += d_vy[i];
}
int main(){

    clock_t start, end;
    double time_used;
    char  str[MAX_CHAR];
    FILE *file;
    int noOfObjects;
    int i;

    file = fopen( DATAFILE , "r");
    fscanf(file,"%s",str);
    noOfObjects = atoi(str);
    printf("Number of objects: %d\n",noOfObjects);

    const int OBJECTS_BYTES = noOfObjects * sizeof(double);

    double *h_x = (double *) malloc(sizeof(double) * noOfObjects);
    double *h_y = (double *) malloc(sizeof(double) * noOfObjects);
    double *h_vx = (double *) malloc(sizeof(double) * noOfObjects);
    double *h_vy = (double *) malloc(sizeof(double) * noOfObjects);
    double *h_m = (double *) malloc(sizeof(double) * noOfObjects);

    double *x0 = (double *) malloc(sizeof(double) * noOfObjects);
    double *y0 = (double *) malloc(sizeof(double) * noOfObjects);
    double *vx0 = (double *) malloc(sizeof(double) * noOfObjects);
    double *vy0 = (double *) malloc(sizeof(double) * noOfObjects);

    double *d_x, *d_y, *d_vx, *d_vy, *d_m;

    hipMalloc((void**) &d_x, OBJECTS_BYTES);
    hipMalloc((void**) &d_y, OBJECTS_BYTES);
    hipMalloc((void**) &d_vx, OBJECTS_BYTES);
    hipMalloc((void**) &d_vy, OBJECTS_BYTES);
    hipMalloc((void**) &d_m, OBJECTS_BYTES);

    printf("\n");

    for (i = 0; i < noOfObjects; i++) {
      fscanf(file, "%s", str);
      h_x[i] = atof(str);
      x0[i] = atof(str);
      fscanf(file, "%s", str);
      h_y[i] = atof(str);
      y0[i] = atof(str);
      fscanf(file, "%s", str);
      h_vx[i] = atof(str);
      vx0[i] = atof(str);
      fscanf(file, "%s", str);
      h_vy[i] = atof(str);
      vy0[i] = atof(str);
      fscanf(file, "%s", str);
      h_m[i] = atof(str);
    }
    fclose(file);

    hipMemcpy(d_x, h_x, OBJECTS_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, OBJECTS_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_vx, h_vx, OBJECTS_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, OBJECTS_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_m, h_m, OBJECTS_BYTES, hipMemcpyHostToDevice);
    start = clock();
    for (int niter = 0;niter < NUM_ITER;niter++) {

      //1024
      //dim3 dimBlock(32, 32);
      //dim3 dimGrid(32, 32);
      //256
      dim3 dimBlock(32, 32);
      dim3 dimGrid(8, 8);
      calculateAsteroid<<<dimGrid, dimBlock>>>(d_x, d_y, d_vx, d_vy, d_m);
      updatePositions<<<1, noOfObjects>>>(d_x, d_y, d_vx, d_vy);

      if (niter % NUM_ITER_SHOW == 0) {
        printf("Iteration %d/%d\n", niter, NUM_ITER);
      }
    }  // nIter
    end = clock();
    hipMemcpy(h_x, d_x, OBJECTS_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, OBJECTS_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(h_vx, d_vx, OBJECTS_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(h_vy, d_vy, OBJECTS_BYTES, hipMemcpyDeviceToHost);
    file = fopen( RESULTSFILE , "w");
    fprintf(file, "Movement of objects\n");
    fprintf(file, "-------------------\n");
    for (i = 0; i < noOfObjects; i++) {
        double mov = sqrt(pow((x0[i] - h_x[i]), 2.0) + pow((y0[i] - h_y[i]), 2.0));
        fprintf(file,"  Object %i  -  %f meters\n", i, mov);
    }
    int hours = NUM_ITER / 3600;
    int mins = (NUM_ITER - hours * 3600) / 60;
    int secs = (NUM_ITER - hours * 3600 - mins * 60);
    fprintf(file,"Time elapsed: %i seconds (%i hours, %i minutes, %i seconds)\n", NUM_ITER, hours, mins, secs);

    time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    fprintf(file,"Processing time: %f sec.\n", time_used);
    fclose(file);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_m);

    free(h_x);
    free(h_y);
    free(h_vx);
    free(h_vy);
    free(h_m);

    return 0;
}  // main